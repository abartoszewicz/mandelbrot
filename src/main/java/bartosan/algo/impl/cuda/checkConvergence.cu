
#include <hip/hip_runtime.h>

extern "C"
__global__ void checkConvergence(float *coordArray, int *result)
{
    float z = 0;
    float zi = 0;
    float ciFloat = coordArray[0];
    float cFloat = coordArray[1];
    int   steps = result[0];

    for (int i = 0; i < steps; i++) {
        float ziT = 2 * (z * zi);
        float zT = z * z - (zi * zi);
        z = zT + cFloat;
        zi = ziT + ciFloat;

        if (z * z + zi * zi >= 4.0) {
                *result = i;
		return;
        }
    }
    *result = steps;
}